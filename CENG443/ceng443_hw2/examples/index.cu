
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void mykernel(void) {
int i = blockDim.x * blockIdx.x + threadIdx.x;
printf("***blockDim=%d\n",blockDim.x);
printf("***blockIdx=%d\n",blockIdx.x);
printf("***threadIdx=%d\n",threadIdx.x);
printf("***index=%d\n",i);
}

int main() {
  mykernel<<<1,1>>>();
  
  return 0;
}
