#include "hip/hip_runtime.h"
#include	"wb.h"

const int BLUR_SIZE = 3;

__global__ void bluringKernel(int imageWidth, int imageHeight, int imageChannels, float* inputImageData, float* outputImageData) {
    
    int col = blockDim.x * blockIdx.x + threadIdx.x;
    int row = blockDim.y * blockIdx.y + threadIdx.y;

    int size = imageWidth * imageHeight * imageChannels;

    if (col < imageWidth && row < imageHeight) {

        float pixVal = 0;
        int pixels = 0;

        for(int blurRow = -BLUR_SIZE; blurRow < BLUR_SIZE+1; ++blurRow) {
            for(int blurCol = -BLUR_SIZE; blurCol < BLUR_SIZE+1; ++blurCol) {
                int curRow = row + blurRow;
                int curCol = col + blurCol;
                // Verify we have a valid image pixel
                if(curRow > -1 && curRow < imageHeight && curCol > -1 && curCol < imageWidth) {
                    pixVal += inputImageData[(curRow * imageWidth + curCol) * imageChannels];
                    pixels++; // Keep track of number of pixels in the accumulated total
                }
            }
        }

        outputImageData[(row * imageWidth + col) * imageChannels] = pixVal / pixels;
    }
}

void wbImage_save(const wbImage_t& image, const char* fName) {

    std::ostringstream oss;
    oss << "P6\n" << "# Created for blurring output " << "\n" << image.width << " " << image.height << "\n" << image.colors << "\n";
    std::string headerStr(oss.str());

    std::ofstream outFile(fName, std::ios::binary);
    outFile.write(headerStr.c_str(), headerStr.size());

    const int numElements = image.width * image.height * image.channels;

    unsigned char* rawData = new unsigned char[numElements];

    for (int i = 0; i < numElements; ++i)
    {
        rawData[i] = static_cast<unsigned char>(image.data[i] * wbInternal::kImageColorLimit + 0.5f);
    }

    outFile.write(reinterpret_cast<char*>(rawData), numElements);
    outFile.close();

    delete [] rawData;
}

int main(int argc, char ** argv) {
    
    char * inputImageFile;
    char * outputImageFile;
    wbImage_t inputImage;
    wbImage_t outputImage;

    float * hostInputImageData;
    float * hostOutputImageData;

    inputImageFile = argv[1];
    outputImageFile = argv[2];
    printf("Loading %s\n", inputImageFile);
    inputImage = wbImport(inputImageFile);
    hostInputImageData = wbImage_getData(inputImage);

    int imageWidth = wbImage_getWidth(inputImage);
    int imageHeight = wbImage_getHeight(inputImage);
    int imageChannels = wbImage_getChannels(inputImage);
    
    printf("%d %d %d\n", imageWidth, imageHeight, imageChannels);
    
    outputImage = wbImage_new(imageWidth, imageHeight, imageChannels);
    hostOutputImageData = wbImage_getData(outputImage);

    /*YOUR CODE FROM HERE*/

    size_t size = imageWidth * imageHeight * imageChannels * sizeof(float);

    float* deviceInputImageData;
    float* deviceOutputImageData;

    hipMalloc((void **) &deviceInputImageData, size);
    hipMalloc((void **) &deviceOutputImageData, size);

    hipMemcpy(deviceInputImageData, hostInputImageData, size, hipMemcpyHostToDevice);

    dim3 dimGrid(1, 1);
    dim3 dimBlock(imageWidth, imageHeight);

    bluringKernel<<<dimGrid, dimBlock>>> (imageWidth, imageHeight, imageChannels, deviceInputImageData, deviceOutputImageData);

    hipMemcpy(hostOutputImageData, deviceOutputImageData, size, hipMemcpyDeviceToHost);
    
    /*UNTIL HERE*/
    
    wbImage_save(outputImage, outputImageFile);    
    return 0;
}


 

