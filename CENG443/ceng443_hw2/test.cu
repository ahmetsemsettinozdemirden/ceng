#include "hip/hip_runtime.h"
#include	"wb.h"

const int BLUR_SIZE = 1; // distance to edge, extual blur size is 3

__global__ void bluringKernel(int imageWidth, int imageHeight, int imageChannels, float* inputImageData, float* outputImageData) {
    
    int col = blockDim.x * blockIdx.x + threadIdx.x;
    int row = blockDim.y * blockIdx.y + threadIdx.y;

    if (col < imageWidth && row < imageHeight) {

        for(int ch = 0; ch < imageChannels; ch++) {

            float data = 0;
            int dataCount = 0;

            for(int blurRow = -BLUR_SIZE; blurRow < BLUR_SIZE+1; ++blurRow) {
                for(int blurCol = -BLUR_SIZE; blurCol < BLUR_SIZE+1; ++blurCol) {
                    int curRow = row + blurRow;
                    int curCol = col + blurCol;
                    // Verify we have a valid image pixel
                    if(curRow > -1 && curRow < imageHeight && curCol > -1 && curCol < imageWidth) {
                            data += inputImageData[(curRow * imageWidth + curCol) * imageChannels + ch];
                            dataCount++; // Keep track of number of pixels(channels) in the accumulated total
                    }
                }
            }
            
            outputImageData[(row * imageWidth + col) * imageChannels + ch] = data / dataCount;
        }
    }
}

void wbImage_save(const wbImage_t& image, const char* fName) {

    std::ostringstream oss;
    oss << "P6\n" << "# Created for blurring output " << "\n" << image.width << " " << image.height << "\n" << image.colors << "\n";
    std::string headerStr(oss.str());

    std::ofstream outFile(fName, std::ios::binary);
    outFile.write(headerStr.c_str(), headerStr.size());

    const int numElements = image.width * image.height * image.channels;

    unsigned char* rawData = new unsigned char[numElements];

    for (int i = 0; i < numElements; ++i)
    {
        rawData[i] = static_cast<unsigned char>(image.data[i] * wbInternal::kImageColorLimit + 0.5f);
    }

    outFile.write(reinterpret_cast<char*>(rawData), numElements);
    outFile.close();

    delete [] rawData;
}

int main(int argc, char ** argv) {
    
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    char * inputImageFile;
    char * outputImageFile;
    wbImage_t inputImage;
    wbImage_t outputImage;

    float * hostInputImageData;
    float * hostOutputImageData;

    inputImageFile = argv[1];
    outputImageFile = argv[2];
    printf("Loading %s\n", inputImageFile);
    inputImage = wbImport(inputImageFile);
    hostInputImageData = wbImage_getData(inputImage);

    int imageWidth = wbImage_getWidth(inputImage);
    int imageHeight = wbImage_getHeight(inputImage);
    int imageChannels = wbImage_getChannels(inputImage);
    
    printf("%d %d %d\n", imageWidth, imageHeight, imageChannels);
    
    outputImage = wbImage_new(imageWidth, imageHeight, imageChannels);
    hostOutputImageData = wbImage_getData(outputImage);

    /*YOUR CODE FROM HERE*/

    int imageSize = imageWidth * imageHeight * imageChannels;
    size_t size = imageSize * sizeof(float);

    float* deviceInputImageData;
    float* deviceOutputImageData;

    err = hipMalloc((void **) &deviceInputImageData, size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate deviceInputImageData (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMalloc((void **) &deviceOutputImageData, size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate deviceInputImageData (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(deviceInputImageData, hostInputImageData, size, hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy inputImageData from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    int threadsPerBlock = 16;
    dim3 dimGrid((imageWidth + threadsPerBlock - 1) / threadsPerBlock, (imageHeight + threadsPerBlock - 1) / threadsPerBlock);
    dim3 dimBlock(threadsPerBlock, threadsPerBlock);

    bluringKernel<<<dimGrid, dimBlock>>> (imageWidth, imageHeight, imageChannels, deviceInputImageData, deviceOutputImageData);
    err = hipGetLastError();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch bluringKernel kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(hostOutputImageData, deviceOutputImageData, size, hipMemcpyDeviceToHost);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy output from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    
    /*UNTIL HERE*/
    
    wbImage_save(outputImage, outputImageFile);  

    return 0;
}


 

