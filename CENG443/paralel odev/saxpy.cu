#include "hip/hip_runtime.h"
/**
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/**
 * Vector addition: C = A + B.
 *
 * This sample is a very basic sample that implements element by element
 * vector addition. It is the same as the sample illustrating Chapter 2
 * of the programming guide with some additions like error checking.
 */

#include "wb.h"
#include <stdio.h>
#include <time.h>

// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>

/**
 * CUDA Kernel Device code
 *
 * Computes the vector addition of A and B into C. The 3 vectors have the same
 * number of elements numElements.
 */
__global__ void
saxpy(const float *A, const float *B, float *C, int scalar, int numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements)
    {
        C[i] = scalar * A[i] + B[i];
    }
}

/**
 * Host main routine
 */
int main(int argc, char ** argv)
{

    int deviceCount;
    wbArg_read(argc, argv);
    hipGetDeviceCount(&deviceCount);
    wbTime_start(GPU, "Getting GPU Data."); //@@ start a timer

    for (int dev = 0; dev < deviceCount; dev++) {
        hipDeviceProp_t deviceProp;

        hipGetDeviceProperties(&deviceProp, dev);

        if (dev == 0) {
            if (deviceProp.major == 9999 && deviceProp.minor == 9999) {
                wbLog(TRACE, "No CUDA GPU has been detected");
                return -1;
            } else if (deviceCount == 1) {
                //@@ WbLog is a provided logging API (similar to Log4J).
                //@@ The logging function wbLog takes a level which is either
                //@@ OFF, FATAL, ERROR, WARN, INFO, DEBUG, or TRACE and a
                //@@ message to be printed.
                wbLog(TRACE, "There is 1 device supporting CUDA");
            } else {
                wbLog(TRACE, "There are ", deviceCount, " devices supporting CUDA");
            }
        }

        wbLog(TRACE, "Device ", dev, " name: ", deviceProp.name);
        wbLog(TRACE, " Computational Capabilities: ", deviceProp.major, ".", deviceProp.minor);
        wbLog(TRACE, " Maximum global memory size: ", deviceProp.totalGlobalMem);
        wbLog(TRACE, " Maximum constant memory size: ", deviceProp.totalConstMem);
        wbLog(TRACE, " Maximum shared memory size per block: ", deviceProp.sharedMemPerBlock);
        wbLog(TRACE, " Maximum block dimensions: ", deviceProp.maxThreadsDim[0], " x ",
                                                    deviceProp.maxThreadsDim[1], " x ",
                                                    deviceProp.maxThreadsDim[2]);
        wbLog(TRACE, " Maximum grid dimensions: ", deviceProp.maxGridSize[0], " x ",
                                                   deviceProp.maxGridSize[1], " x ",
                                                   deviceProp.maxGridSize[2]);
        wbLog(TRACE, " Warp size: ", deviceProp.warpSize);
    }
    wbTime_stop(GPU, "Getting GPU Data."); //@@ stop the timer

    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;


    if(argc != 3) {
        printf("usage: ./saxpy <size of the arrays> <scalar value>");
        return -1;
    }

    // Print the vector length to be used, and compute its size
    clock_t begin = clock();
    int numElements = atoi(argv[1]);
    int scalar = atoi(argv[2]);
    size_t size = numElements * sizeof(float);
    printf("[Saxpy of %d elements with scalar value of %d]\n", numElements, scalar);

    // Allocate the host input vector A
    float *h_A = (float *)malloc(size);

    // Allocate the host input vector B
    float *h_B = (float *)malloc(size);

    // Allocate the host output vector C
    float *h_C = (float *)malloc(size);

    // Verify that allocations succeeded
    if (h_A == NULL || h_B == NULL || h_C == NULL)
    {
        fprintf(stderr, "Failed to allocate host vectors!\n");
        exit(EXIT_FAILURE);
    }

    // Initialize the host input vectors
    for (int i = 0; i < numElements; ++i)
    {
        h_A[i] = rand()/(float)RAND_MAX;
        h_B[i] = rand()/(float)RAND_MAX;
    }

    // Allocate the device input vector A
    float *d_A = NULL;
    err = hipMalloc((void **)&d_A, size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Allocate the device input vector B
    float *d_B = NULL;
    err = hipMalloc((void **)&d_B, size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector B (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Allocate the device output vector C
    float *d_C = NULL;
    err = hipMalloc((void **)&d_C, size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector C (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy the host input vectors A and B in host memory to the device input vectors in
    // device memory
    printf("Copy input data from the host memory to the CUDA device\n");
    err = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector A from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector B from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Launch the Vector Add CUDA Kernel
    int threadsPerBlock = 256;
    int blocksPerGrid =(numElements + threadsPerBlock - 1) / threadsPerBlock;
    printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
    saxpy<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, scalar, numElements);
    err = hipGetLastError();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch saxpy kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy the device result vector in device memory to the host result vector
    // in host memory.
    printf("Copy output data from the CUDA device to the host memory\n");
    err = hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector C from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Verify that the result vector is correct
    for (int i = 0; i < numElements; ++i)
    {
        if (fabs(scalar * h_A[i] + h_B[i] - h_C[i]) > 1e-5)
        {
            fprintf(stderr, "Result verification failed at element %d!\n", i);
            exit(EXIT_FAILURE);
        }
    }

    printf("Test PASSED\n");

    // Free device global memory
    err = hipFree(d_A);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_B);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector B (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_C);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector C (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Free host memory
    free(h_A);
    free(h_B);
    free(h_C);

    clock_t end = clock();
    printf("Done. completed in %d\n", (double)(end - begin) / CLOCKS_PER_SEC);
    return 0;
}

